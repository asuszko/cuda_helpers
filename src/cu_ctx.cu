#include <cassert>
#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_ctx.h"

#include <iostream>
using namespace std;

/**
 *  Initialize a CUDA context on a device.
 *  @param device - [int] : CUDA device.
 *  @return ctx - [hipCtx_t*] : New floating CUDA context handle.
 */
hipCtx_t *cu_context_create(int device)
{
    hipCtx_t *ctx = new hipCtx_t;
    
    int num_devices = 0;
    gpuErrchk(hipGetDeviceCount(&num_devices));
    assert(device < num_devices && "Invalid device_id in cu_create_context");

    gpuContextErrchk(hipCtxCreate(ctx,
                                 hipDeviceMapHost,
                                 device));

    return ctx;
}

/**
 *  Push a CUDA context to the current thread stack.
 *  @param ctx - [hipCtx_t*] : Floating CUDA context handle.
 */
void cu_context_push(hipCtx_t *ctx)
{
    /* Pop the existing context if it exists, then push the new one,
    otherwise, push the context. */
    hipCtx_t tmp_ctx;
    gpuContextErrchk(hipCtxGetCurrent(&tmp_ctx));
    if(tmp_ctx != NULL) {
        cu_context_pop(&tmp_ctx);
    }
    gpuContextErrchk(hipCtxPushCurrent(*ctx));
    return;
}

/**
 *  Pop a CUDA context from the current thread stack.
 *  @param ctx - [hipCtx_t*] : Floating CUDA context handle.
 */
void cu_context_pop(hipCtx_t *ctx)
{
    gpuContextErrchk(hipCtxPopCurrent(ctx));
    return;
}

/**
 *  Destroy a CUDA context, implicitly freeing all resources associated with it.
 *  @param ctx - [hipCtx_t*] : Floating CUDA context handle.
 */
void cu_context_destroy(hipCtx_t *ctx)
{
    gpuContextErrchk(hipCtxDestroy(*ctx));
    delete[] ctx;
    return;
}
