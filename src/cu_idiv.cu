#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <tuple>

#include "cu_idiv.h"
#include "cu_errchk.h"


template <typename T>
__global__ void div1_val(T *y, const T *x, unsigned long long N)
{
    unsigned long long index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long stride = gridDim.x * blockDim.x;
    
    for(; index < N; index += stride) {
        y[index] /= x[0];
    }
}


template <typename T>
__global__ void div1_vec(T *y, const T *x, unsigned long long N)
{
    unsigned long long index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long stride = gridDim.x * blockDim.x;
    
    for(; index < N; index += stride) {
        y[index] /= x[index];
    }     
}


template <typename T>
__global__ void div2_val(T *y, const T *x, unsigned long long N)
{
    unsigned long long index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long stride = gridDim.x * blockDim.x;
    
    for(; index < N; index += stride) {
        T valy = y[index];
        T valx = x[0];
        
        y[index].x = (valy.x*valx.x+valy.y*valx.y)/(valx.x*valx.x+valx.y*valx.y);
        y[index].y = (valy.y*valx.x-valy.x*valx.y)/(valx.x*valx.x+valx.y*valx.y);
    }     
}


template <typename T>
__global__ void div2_vec(T *y, const T *x, unsigned long long N)
{
    unsigned long long index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long stride = gridDim.x * blockDim.x;
    
    for(; index < N; index += stride) {
        T valy = y[index];
        T valx = x[index];
        
        y[index].x = (valy.x*valx.x+valy.y*valx.y)/(valx.x*valx.x+valx.y*valx.y);
        y[index].y = (valy.y*valx.x-valy.x*valx.y)/(valx.x*valx.x+valx.y*valx.y);
    }     
}


void cu_idiv(void *y, void *x, unsigned long long N,
             const int dtype, int dtype_len, bool vec,
             hipStream_t *stream)
{
    dim3 blockSize(256);
    dim3 gridSize((((N-1)/blockSize.x+1)-1)/blockSize.x+1);
    
    hipStream_t stream_id;
    (stream == NULL) ? stream_id = NULL : stream_id = *stream;

    switch(dtype) {
        case 0:
            switch(dtype_len) {
				     case 1:
				         if (vec) div1_vec<<<gridSize,blockSize,0,stream_id>>>((float*)y, (const float*)x, N);
				         else     div1_val<<<gridSize,blockSize,0,stream_id>>>((float*)y, (const float*)x, N);
				         break;
                case 2:
				         if (vec) div2_vec<<<gridSize,blockSize,0,stream_id>>>((float2*)y,(const float2*)x,N);
				         else     div2_val<<<gridSize,blockSize,0,stream_id>>>((float2*)y,(const float2*)x,N);
				         break;
        }
            break;
        case 1:
            switch(dtype_len) {
				     case 1:
				         if (vec) div1_vec<<<gridSize,blockSize,0,stream_id>>>((double*)y, (const double*)x, N);
				         else     div1_val<<<gridSize,blockSize,0,stream_id>>>((double*)y, (const double*)x, N);
				         break;
                case 2:
				         if (vec) div2_vec<<<gridSize,blockSize,0,stream_id>>>((double2*)y,(const double2*)x,N);
				         else     div2_val<<<gridSize,blockSize,0,stream_id>>>((double2*)y,(const double2*)x,N);
				         break;
            }
            break;
    }
}