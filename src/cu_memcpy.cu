#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_memcpy.h"


/**
*  Copy a chunk of memory from the host to device.
*  @param d_arr - [void*] : Pointer to device memory.
*  @param h_arr - [void*] : Pointer to host memory.
*  @param size - [size_t] : Size of the transfer in bytes.
*/
void cu_memcpy_h2d(void *d_arr,
                   void *h_arr,
                   size_t size)
{
    gpuErrchk(hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice));
    return;
}

/**
*  Async Copy a chunk of memory from the host to device.
*  @param d_arr - [void*] : Pointer to device memory.
*  @param h_arr - [void*] : Pointer to host memory.
*  @param size - [size_t] : Size of the transfer in bytes.
*  @param stream - [hipStream_t*] : CUDA stream
*/
void cu_memcpy_h2d_async(void *d_arr,
                         void *h_arr,
                         size_t size,
                         hipStream_t *stream)
{
    gpuErrchk(hipMemcpyAsync(d_arr, h_arr, size, hipMemcpyHostToDevice, *stream));
    return;
}

/**
*  Copy a chunk of memory from the device to host.
*  @param d_arr - [void*] : Pointer to device memory.
*  @param h_arr - [void*] : Pointer to host memory.
*  @param size - [size_t] : Size of the transfer in bytes.
*/
void cu_memcpy_d2h(void *d_arr,
                   void *h_arr,
                   size_t size)
{
    gpuErrchk(hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost));
    return;
}

/**
*  Async copy a chunk of memory from the device to host.
*  @param d_arr - [void*] : Pointer to device memory.
*  @param h_arr - [void*] : Pointer to host memory.
*  @param size - [size_t] : Size of the transfer in bytes.
*  @param stream - [hipStream_t*] : CUDA stream
*/
void cu_memcpy_d2h_async(void *d_arr,
                         void *h_arr,
                         size_t size,
                         hipStream_t *stream)
{
    gpuErrchk(hipMemcpyAsync(h_arr, d_arr, size, hipMemcpyDeviceToHost, *stream));
    return;
}

/**
*  Copy a chunk of memory from the device to device. This copy can be
*  from one device to another, or to another memory space on the same
*  device.
*  @param d_src - [void*] : Pointer to device source memory.
*  @param d_dst - [void*] : Pointer to device destination memory.
*  @param size - [size_t] : Size of the transfer in bytes.
*/
void cu_memcpy_d2d(void *d_src,
                   void *d_dst,
                   size_t size)
{
    gpuErrchk(hipMemcpy(d_dst, d_src, size, hipMemcpyDeviceToDevice));
    return;
}

/**
*  Async copy a chunk of memory from the device to device. This copy can
*  be from one device to another, or to another memory space on the same
*  device.
*  @param d_src - [void*] : Pointer to device source memory.
*  @param d_dst - [void*] : Pointer to device destination memory.
*  @param size - [size_t] : Size of the transfer in bytes.
*  @param stream - [hipStream_t*] : CUDA stream
*/
void cu_memcpy_d2d_async(void *d_src,
                         void *d_dst,
                         size_t size,
                         hipStream_t *stream)
{
    gpuErrchk(hipMemcpyAsync(d_dst, d_src, size, hipMemcpyDeviceToDevice, *stream));
    return;
}

/**
*  Set the byte value of the memory on the device.
*  @param d_arr - [void*] : Pointer to device memory.
*  @param value - [int] : Value to set.
*  @param size - [size_t] : Size in bytes to set.
*/
void cu_memset(void *d_arr,
               int value,
               size_t size)
{
    gpuErrchk(hipMemset(d_arr, value, size));
    return;
}

/**
*  Async set the byte value of the memory on the device.
*  @param d_arr - [void*] : Pointer to device memory.
*  @param value - [int] : Value to set.
*  @param size - [size_t] : Size in bytes to set.
*  @param stream - [hipStream_t*] : CUDA stream
*/
void cu_memset_async(void *d_arr,
                     int value,
                     size_t size,
                     hipStream_t *stream)
{
    gpuErrchk(hipMemsetAsync(d_arr, value, size, *stream));
    return;
}

/**
*  Pin host memory space so that it works with CUDA streams.
*  @param h_arr - [void*] : Pointer to host memory.
*  @param size - [size_t] : Size in bytes to pin.
*/
void cu_mempin(void *h_arr, size_t size)
{
    /* Check if array is already pinned. */
    hipPointerAttribute_t ptr_attr;
    bool is_pinned = (hipPointerGetAttributes(&ptr_attr, h_arr) != hipErrorInvalidValue);

    /* If array is not already pinned, clear out the error and pin it. Else, do nothing. */
    if (!is_pinned) {
        hipGetLastError();
        gpuErrchk(hipHostRegister(h_arr, size, hipHostRegisterPortable));
    }
    return;
}

/**
*  Unpin host memory space.
*  @param h_arr - [void*] : Pointer to host memory.
*/
void cu_memunpin(void *h_arr)
{
    /* Check if array is pinned. */
    hipPointerAttribute_t ptr_attr;
    bool is_pinned = (hipPointerGetAttributes(&ptr_attr, h_arr) != hipErrorInvalidValue);

    if (is_pinned) {
    		gpuErrchk(hipDeviceSynchronize());
    		gpuErrchk(hipHostUnregister(h_arr));
    }
    else {
        hipGetLastError();
    }
    return;
}

/**
*  Create and return the copyparms3D object that is used by a
*  subsequent hipMemcpy3D call.
*  @param src_Array - [void*] : Pointer to host memory.
*  @param dst_Array - [hipArray*] : Pointer to device hipArray.
*  @param extent - [hipExtent] : Dimensions of 3D memory [width,height,depth].
*  @param element_size_bytes [unsigned int] : Size of each element in bytes.
*/
hipMemcpy3DParms cu_copyparams(void *src_Array,
                                hipArray *dst_Array,
                                hipExtent extent,
                                unsigned int element_size_bytes)
{
    int nx = extent.width;
    int ny = extent.height;
    int nz = extent.depth;

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPos   = make_hipPos(0, 0, 0);
    copyParams.dstPos   = make_hipPos(0, 0, 0);
    copyParams.srcPtr   = make_hipPitchedPtr(src_Array, nx*element_size_bytes, nx, ny);
    copyParams.dstArray = dst_Array;
    copyParams.extent   = make_hipExtent(nx, ny, nz);
    copyParams.kind     = hipMemcpyHostToDevice;

    return copyParams;
}

/**
*  Copy a host array to a 3D hipArray.
*  @param src_Array - [void*] : Pointer to host memory.
*  @param dst_Array - [hipArray*] : Pointer to device hipArray.
*  @param extent - [dim3] : Dimensions of 3D memory [x,y,z].
*  @param element_size_bytes [unsigned int] : Size of each element in bytes.
*/
void cu_memcpy_3d(void *src_Array,
                  hipArray *dst_Array,
                  dim3 extent,
                  unsigned int element_size_bytes)
{
    int nx = extent.x;
    int ny = extent.y;
    int nz = extent.z;

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPos   = make_hipPos(0, 0, 0);
    copyParams.dstPos   = make_hipPos(0, 0, 0);
    copyParams.srcPtr   = make_hipPitchedPtr(src_Array, nx*element_size_bytes, nx, ny);
    copyParams.dstArray = dst_Array;
    copyParams.extent   = make_hipExtent(nx, ny, nz);
    copyParams.kind     = hipMemcpyHostToDevice;

    gpuErrchk(hipMemcpy3D(&copyParams));

    return;
}

/**
*  Async Copy a host array to a 3D hipArray.
*  @param src_Array - [void*] : Pointer to host memory.
*  @param dst_Array - [hipArray*] : Pointer to device hipArray.
*  @param extent - [dim3] : Dimensions of 3D memory [x,y,z].
*  @param element_size_bytes [unsigned int] : Size of each element in bytes.
*  @param stream - [hipStream_t*] : CUDA stream
*/
void cu_memcpy_3d_async(void *src_Array,
                        hipArray *dst_Array,
                        dim3 extent,
                        unsigned int element_size_bytes,
                        hipStream_t *stream)
{
    int nx = extent.x;
    int ny = extent.y;
    int nz = extent.z;

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPos   = make_hipPos(0, 0, 0);
    copyParams.dstPos   = make_hipPos(0, 0, 0);
    copyParams.srcPtr   = make_hipPitchedPtr(src_Array, nx*element_size_bytes, nx, ny);
    copyParams.dstArray = dst_Array;
    copyParams.extent   = make_hipExtent(nx, ny, nz);
    copyParams.kind     = hipMemcpyHostToDevice;

    gpuErrchk(hipMemcpy3DAsync(&copyParams,*stream));

    return;
}
