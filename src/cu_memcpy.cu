#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_memcpy.h"


void cu_memcpy_h2d(void *d_arr, void *h_arr, size_t size)
{
	  gpuErrchk(hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice));
		return;
}


void cu_memcpy_h2d_async(void *d_arr, void *h_arr, size_t size,
									       hipStream_t *stream)
{
	  gpuErrchk(hipMemcpyAsync(d_arr, h_arr, size, hipMemcpyHostToDevice, *stream));
		return;
}


void cu_memcpy_d2h(void *d_arr, void *h_arr, size_t size)
{
	  gpuErrchk(hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost));
		return;
}


void cu_memcpy_d2h_async(void *d_arr, void *h_arr, size_t size,
										     hipStream_t *stream)
{
	  gpuErrchk(hipMemcpyAsync(h_arr, d_arr, size, hipMemcpyDeviceToHost, *stream));
		return;
}


void cu_memcpy_d2d(void *d_arr_src, void *d_arr_dst, size_t size)
{
	  gpuErrchk(hipMemcpy(d_arr_dst, d_arr_src, size, hipMemcpyDeviceToDevice));
		return;
}


void cu_memcpy_d2d_async(void *d_arr_src, void *d_arr_dst, size_t size,
												 hipStream_t *stream)
{
	  gpuErrchk(hipMemcpyAsync(d_arr_dst, d_arr_src, size, hipMemcpyDeviceToDevice, *stream));
		return;
}


void cu_memset(void *d_arr, int value, size_t size)
{
	  gpuErrchk(hipMemset(d_arr, value, size));
    return;
}


void cu_memset_async(void *d_arr, int value, size_t size, hipStream_t *stream)
{
	  gpuErrchk(hipMemsetAsync(d_arr, value, size, *stream));
    return;
}


void cu_mempin(void *h_arr, size_t size)
{
    /* Check if array is already pinned. */
    hipPointerAttribute_t ptr_attr;
    bool is_pinned = (hipPointerGetAttributes(&ptr_attr, h_arr) != hipErrorInvalidValue);

    /* If array is not already pinned, clear out the error and pin it. Else, do nothing. */
    if (!is_pinned) {
        hipGetLastError();
        gpuErrchk(hipHostRegister(h_arr, size, hipHostRegisterPortable));
    }
    return;
}


void cu_memunpin(void *h_arr)
{
    /* Check if array is pinned. */
    hipPointerAttribute_t ptr_attr;
    bool is_pinned = (hipPointerGetAttributes(&ptr_attr, h_arr) != hipErrorInvalidValue);

    if (is_pinned) {
    		gpuErrchk(hipDeviceSynchronize());
    		gpuErrchk(hipHostUnregister(h_arr));
    }
    else {
        hipGetLastError();
    }
    return;
}


hipMemcpy3DParms cu_copyparams(void* src_Array,
							                  hipArray* dst_Array,
							                  hipExtent extent,
							                  unsigned int element_size_bytes)
{
	  int nx = extent.width;
	  int ny = extent.height;
	  int nz = extent.depth;


		hipMemcpy3DParms copyParams = {0};

		copyParams.srcPos   = make_hipPos(0, 0, 0);
		copyParams.dstPos   = make_hipPos(0, 0, 0);
		copyParams.srcPtr   = make_hipPitchedPtr(src_Array, nx*element_size_bytes, nx, ny);
		copyParams.dstArray = dst_Array;
		copyParams.extent   = make_hipExtent(nx, ny, nz);
		copyParams.kind     = hipMemcpyHostToDevice;

		return copyParams;
}



// void cu_memcpy3d(void* src_Array,
//                  hipArray* dst_Array,
//                  hipExtent extent,
//                  unsigned int element_size_bytes)
// {
//     int nx = extent.width;
//     int ny = extent.height;
//     int nz = extent.depth;
//
//     hipMemcpy3DParms copyParams = {0};
//
//     copyParams.srcPos   = make_hipPos(0, 0, 0);
//   	copyParams.dstPos   = make_hipPos(0, 0, 0);
//   	copyParams.srcPtr   = make_hipPitchedPtr(src_Array, nx*element_size_bytes, nx, ny);
//     copyParams.dstArray = dst_Array;
//     copyParams.extent   = make_hipExtent(nx, ny, nz);
//     copyParams.kind     = hipMemcpyHostToDevice;
//
//     gpuErrchk(hipMemcpy3D(&copyParams));
//
//     return;
// }
//
//
void cu_memcpy_3d_async(void *src_Array,
                        hipArray *dst_Array,
                        dim3 extent,
                        unsigned int element_size_bytes,
                        hipStream_t *stream)
{
    int nx = extent.x;
    int ny = extent.y;
    int nz = extent.z;

    hipMemcpy3DParms copyParams = {0};

    copyParams.srcPos   = make_hipPos(0, 0, 0);
  	copyParams.dstPos   = make_hipPos(0, 0, 0);
  	copyParams.srcPtr   = make_hipPitchedPtr(src_Array, nx*element_size_bytes, nx, ny);
    copyParams.dstArray = dst_Array;
    copyParams.extent   = make_hipExtent(nx, ny, nz);
    copyParams.kind     = hipMemcpyHostToDevice;

    gpuErrchk(hipMemcpy3DAsync(&copyParams,*stream));

    return;
}
