#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_channel.h"

/**
*  Create a char hipChannelFormatDesc object. CUDA has built-in support 
*  for vector types: multi-dimensional data with 1 to 4 components, 
*  addressed by x,y,x,w.
*  @param ncomponents - [int] : Number of components in the channel.
*  @param is_unsigned - [bool] : Unsigned flag.
*  @return channelDescData - [hipChannelFormatDesc] : char hipChannelFormatDesc object.
*/
hipChannelFormatDesc cu_create_channel_char(int ncomponents,
                                             bool is_unsigned)
{
    hipChannelFormatDesc channelDescData;
    if(is_unsigned) {
        switch(ncomponents) {
            case 1: {
                channelDescData = hipCreateChannelDesc<unsigned char>();
                break;
            }

            case 2: {
                channelDescData = hipCreateChannelDesc<uchar2>();
                break;
            }

            case 3: {
                channelDescData = hipCreateChannelDesc<uchar3>();
                break;
            }

            case 4: {
                channelDescData = hipCreateChannelDesc<uchar4>();
                break;
            }
        }
    }
    else {
        switch(ncomponents) {
            case 1: {
                channelDescData = hipCreateChannelDesc<char>();
                break;
            }

            case 2: {
                channelDescData = hipCreateChannelDesc<char2>();
                break;
            }

            case 3: {
                channelDescData = hipCreateChannelDesc<char3>();
                break;
            }

            case 4: {
                channelDescData = hipCreateChannelDesc<char4>();
                break;
            }
        }
    }
    return channelDescData;
}

/**
*  Create a short hipChannelFormatDesc object. CUDA has built-in support 
*  for vector types: multi-dimensional data with 1 to 4 components, 
*  addressed by x,y,x,w.
*  @param ncomponents - [int] : Number of components in the channel.
*  @param is_unsigned - [bool] : Unsigned flag.
*  @return channelDescData - [hipChannelFormatDesc] : short hipChannelFormatDesc object.
*/
hipChannelFormatDesc cu_create_channel_short(int ncomponents,
                                              bool is_unsigned)
{
    hipChannelFormatDesc channelDescData;
    if(is_unsigned) {
        switch(ncomponents) {
            case 1: {
                channelDescData = hipCreateChannelDesc<unsigned short>();
                break;
            }

            case 2: {
                channelDescData = hipCreateChannelDesc<ushort2>();
                break;
            }

            case 3: {
                channelDescData = hipCreateChannelDesc<ushort3>();
                break;
            }

            case 4: {
                channelDescData = hipCreateChannelDesc<ushort4>();
                break;
            }
        }
    }
    else {
        switch(ncomponents) {
            case 1: {
                channelDescData = hipCreateChannelDesc<short>();
                break;
            }

            case 2: {
                channelDescData = hipCreateChannelDesc<short2>();
                break;
            }

            case 3: {
                channelDescData = hipCreateChannelDesc<short3>();
                break;
            }

            case 4: {
                channelDescData = hipCreateChannelDesc<short4>();
                break;
            }
        }
    }
    return channelDescData;
}

/**
*  Create a half hipChannelFormatDesc object. CUDA has built-in support 
*  for vector types: multi-dimensional data with 1 to 4 components, 
*  addressed by x,y,x,w.
*  @return channelDescData - [hipChannelFormatDesc] : half hipChannelFormatDesc object.
*/
hipChannelFormatDesc cu_create_channel_half()
{
    hipChannelFormatDesc channelDescData;
    channelDescData = cudaCreateChannelDescHalf();
    return channelDescData;
}

/**
*  Create a float hipChannelFormatDesc object. CUDA has built-in support 
*  for vector types: multi-dimensional data with 1 to 4 components, 
*  addressed by x,y,x,w.
*  @param ncomponents - [int] : Number of components in the channel.
*  @return channelDescData - [hipChannelFormatDesc] : float hipChannelFormatDesc object.
*/
hipChannelFormatDesc cu_create_channel_float(int ncomponents)
{
    hipChannelFormatDesc channelDescData;
    switch(ncomponents) {
        case 1: {
            channelDescData = hipCreateChannelDesc<float>();
            break;
        }

        case 2: {
            channelDescData = hipCreateChannelDesc<float2>();
            break;
        }

        case 3: {
            channelDescData = hipCreateChannelDesc<float3>();
            break;
        }

        case 4: {
            channelDescData = hipCreateChannelDesc<float4>();
            break;
        }
    }
    return channelDescData;
}