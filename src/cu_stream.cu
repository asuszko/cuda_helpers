#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_stream.h"


hipStream_t *cu_stream_create()
{
    hipStream_t *stream = (hipStream_t*) malloc(sizeof(hipStream_t));
    gpuErrchk(hipStreamCreate(stream));
    return stream;
}


void cu_stream_destroy(hipStream_t *stream)
{
    gpuErrchk(hipStreamSynchronize(*stream));
    gpuErrchk(hipStreamDestroy(*stream));
    free(stream);
    return;
}
