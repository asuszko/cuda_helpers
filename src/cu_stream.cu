#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_stream.h"


/**
*  Create a CUDA stream.
*  @return stream - [hipStream_t*] - CUDA stream
*/
hipStream_t *cu_stream_create()
{
    hipStream_t *stream = new hipStream_t;
    gpuErrchk(hipStreamCreate(stream));
    return stream;
}

/**
*  Destroy a CUDA stream.
*  @param stream - [hipStream_t*] - CUDA stream
*/
void cu_stream_destroy(hipStream_t *stream)
{
    gpuErrchk(hipStreamSynchronize(*stream));
    gpuErrchk(hipStreamDestroy(*stream));
    delete[] stream;
    return;
}
