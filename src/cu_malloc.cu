#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_malloc.h"


void* cu_malloc(size_t size)
{
	  void* d_arr;
    gpuErrchk(hipMalloc((void **)&d_arr, size));
	  return d_arr;
}


void *cu_malloc_managed(size_t size)
{
    void *arr;
    gpuErrchk(hipMallocManaged(&arr, size));
    return arr;
}


hipArray *cu_malloc_3d(hipChannelFormatDesc *channel,
											  dim3 extent,
											  bool layered)
{
    hipArray *cu_array;
		if (layered) {
		    gpuErrchk(hipMalloc3DArray(&cu_array,
																		channel,
																		make_hipExtent(extent.x, extent.y, extent.z),
																		hipArrayLayered));
		}
		else {
				gpuErrchk(hipMalloc3DArray(&cu_array,
																		channel,
																		make_hipExtent(extent.x, extent.y, extent.z)));
		}

    return cu_array;
}
