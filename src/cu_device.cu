#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_device.h"

/**
*  Get the hipGetDeviceProperties object.
*  @param device - [int] : Device id to query.
*  @return props - [hipDeviceProp_t] : hipDeviceProp_t object.
*/
hipDeviceProp_t cu_device_props(int device)
{
    hipDeviceProp_t props;
    gpuErrchk(hipGetDeviceProperties(&props,device));
    return props;
}

/**
*  Get the number of CUDA devices.
*  @return count - [int] : Number of CUDA devices.
*/
int cu_device_count()
{
    int count;
    gpuErrchk(hipGetDeviceCount(&count));
    return count;
}

/**
*  Reset the device on the current CUDA context.
*/
void cu_device_reset()
{
    gpuErrchk(hipDeviceReset());
    return;
}

/**
* Get memory info of the device.
*/
void cu_get_mem_info(size_t *free, size_t *total)
{
    gpuErrchk(hipMemGetInfo(free, total));
    return;	
}