#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_device.h"


hipDeviceProp_t cu_device_props(int device)
{
    hipDeviceProp_t props;
    gpuErrchk(hipGetDeviceProperties(&props,device));
    return props;
}


int cu_device_count()
{
    int count;
    gpuErrchk(hipGetDeviceCount(&count));
    return count;
}


void cu_device_reset()
{
	  gpuErrchk(hipDeviceReset());
    return;
}
