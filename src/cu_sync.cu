#include <hip/hip_runtime.h>
#include "cu_errchk.h"
#include "cu_sync.h"


/**
*  Block host thread until the CUDA device finishes all tasks.
*/
void cu_sync_device()
{
    gpuErrchk(hipDeviceSynchronize());
    return;
}


/**
*  Block host thread until the CUDA stream finishes all tasks.
*  @param stream - [hipStream_t*] - CUDA stream handle
*/
void cu_sync_stream(hipStream_t *stream)
{
    gpuErrchk(hipStreamSynchronize(*stream));
    return;
}
